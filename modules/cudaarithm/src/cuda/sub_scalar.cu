/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <typename T, typename S, typename D> struct SubScalar : unary_function<T, D>
    {
        S val;
        int scale;

        __host__ SubScalar(S val_, int scale_) : val(val_), scale(scale_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(scale * (a - val));
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::SubScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void subScalar(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        SubScalar<T, S, D> op(static_cast<S>(val), inv ? -1 : 1);

        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void subScalar<uchar, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<uchar, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void subScalar<schar, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<schar, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<ushort, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<ushort, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<ushort, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<short, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<short, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<short, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<int, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<int, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<int, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<float, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<float, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<float, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<float, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subScalar<double, double, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subScalar<double, double, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subScalar<double, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
}

#endif // CUDA_DISABLER
