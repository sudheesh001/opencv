/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct VSub4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub4(a, b);
        }

        __host__ __device__ __forceinline__ VSub4() {}
        __host__ __device__ __forceinline__ VSub4(const VSub4&) {}
    };

    struct VSub2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub2(a, b);
        }

        __host__ __device__ __forceinline__ VSub2() {}
        __host__ __device__ __forceinline__ VSub2(const VSub2&) {}
    };

    template <typename T, typename D> struct SubMat : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a - b);
        }

        __host__ __device__ __forceinline__ SubMat() {}
        __host__ __device__ __forceinline__ SubMat(const SubMat&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VSub4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <> struct TransformFunctorTraits< arithm::VSub2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::SubMat<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void subMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VSub4(), WithOutMask(), stream);
    }

    void subMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VSub2(), WithOutMask(), stream);
    }

    template <typename T, typename D>
    void subMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, SubMat<T, D>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, SubMat<T, D>(), WithOutMask(), stream);
    }

    template void subMat<uchar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<uchar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void subMat<schar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<schar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<ushort, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<ushort, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<ushort, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<short, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<short, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<short, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<int, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<int, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<int, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<float, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void subMat<double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void subMat<double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void subMat<double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
}

#endif // CUDA_DISABLER
