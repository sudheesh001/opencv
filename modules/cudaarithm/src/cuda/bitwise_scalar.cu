/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace cv { namespace cuda { namespace device
{
    template <typename T> struct TransformFunctorTraits< binder2nd< bit_and<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< bit_or<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< bit_xor<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T> void bitScalarAnd(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::cuda::device::bind2nd(bit_and<T>(), src2), WithOutMask(), stream);
    }

    template <typename T> void bitScalarOr(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::cuda::device::bind2nd(bit_or<T>(), src2), WithOutMask(), stream);
    }

    template <typename T> void bitScalarXor(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::cuda::device::bind2nd(bit_xor<T>(), src2), WithOutMask(), stream);
    }

    template void bitScalarAnd<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarAnd<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarAnd<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarAnd<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);

    template void bitScalarOr<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarOr<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarOr<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarOr<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);

    template void bitScalarXor<uchar>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarXor<ushort>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarXor<int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
    template void bitScalarXor<unsigned int>(PtrStepSzb src1, uint src2, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
