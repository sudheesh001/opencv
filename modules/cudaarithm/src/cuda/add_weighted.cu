/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <typename T> struct UseDouble_
    {
        enum {value = 0};
    };
    template <> struct UseDouble_<double>
    {
        enum {value = 1};
    };
    template <typename T1, typename T2, typename D> struct UseDouble
    {
        enum {value = (UseDouble_<T1>::value || UseDouble_<T2>::value || UseDouble_<D>::value)};
    };

    template <typename T1, typename T2, typename D, bool useDouble> struct AddWeighted_;
    template <typename T1, typename T2, typename D> struct AddWeighted_<T1, T2, D, false> : binary_function<T1, T2, D>
    {
        float alpha;
        float beta;
        float gamma;

        __host__ AddWeighted_(double alpha_, double beta_, double gamma_) : alpha(static_cast<float>(alpha_)), beta(static_cast<float>(beta_)), gamma(static_cast<float>(gamma_)) {}

        __device__ __forceinline__ D operator ()(T1 a, T2 b) const
        {
            return saturate_cast<D>(a * alpha + b * beta + gamma);
        }
    };
    template <typename T1, typename T2, typename D> struct AddWeighted_<T1, T2, D, true> : binary_function<T1, T2, D>
    {
        double alpha;
        double beta;
        double gamma;

        __host__ AddWeighted_(double alpha_, double beta_, double gamma_) : alpha(alpha_), beta(beta_), gamma(gamma_) {}

        __device__ __forceinline__ D operator ()(T1 a, T2 b) const
        {
            return saturate_cast<D>(a * alpha + b * beta + gamma);
        }
    };
    template <typename T1, typename T2, typename D> struct AddWeighted : AddWeighted_<T1, T2, D, UseDouble<T1, T2, D>::value>
    {
        AddWeighted(double alpha_, double beta_, double gamma_) : AddWeighted_<T1, T2, D, UseDouble<T1, T2, D>::value>(alpha_, beta_, gamma_) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <typename T1, typename T2, typename D, size_t src1_size, size_t src2_size, size_t dst_size> struct AddWeightedTraits : DefaultTransformFunctorTraits< arithm::AddWeighted<T1, T2, D> >
    {
    };
    template <typename T1, typename T2, typename D, size_t src_size, size_t dst_size> struct AddWeightedTraits<T1, T2, D, src_size, src_size, dst_size> : arithm::ArithmFuncTraits<src_size, dst_size>
    {
    };

    template <typename T1, typename T2, typename D> struct TransformFunctorTraits< arithm::AddWeighted<T1, T2, D> > : AddWeightedTraits<T1, T2, D, sizeof(T1), sizeof(T2), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T1, typename T2, typename D>
    void addWeighted(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream)
    {
        AddWeighted<T1, T2, D> op(alpha, beta, gamma);

        device::transform((PtrStepSz<T1>) src1, (PtrStepSz<T2>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void addWeighted<uchar, uchar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, uchar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, schar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, schar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<uchar, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<uchar, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<schar, schar, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, schar, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<schar, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<schar, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<ushort, ushort, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, ushort, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<ushort, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<ushort, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<short, short, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, short, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<short, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<short, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<int, int, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, int, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<int, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<int, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<int, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<float, float, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, float, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);

    template void addWeighted<float, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<float, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);



    template void addWeighted<double, double, uchar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, schar>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, ushort>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, short>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, int>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, float>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
    template void addWeighted<double, double, double>(PtrStepSzb src1, double alpha, PtrStepSzb src2, double beta, double gamma, PtrStepSzb dst, hipStream_t stream);
}

#endif /* CUDA_DISABLER */
