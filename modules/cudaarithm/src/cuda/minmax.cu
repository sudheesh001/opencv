#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/utility.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace minMax
{
    __device__ unsigned int blocks_finished = 0;

    // To avoid shared bank conflicts we convert each value into value of
    // appropriate type (32 bits minimum)
    template <typename T> struct MinMaxTypeTraits;
    template <> struct MinMaxTypeTraits<uchar> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<schar> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<ushort> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<int> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<float> { typedef float best_type; };
    template <> struct MinMaxTypeTraits<double> { typedef double best_type; };

    template <int BLOCK_SIZE, typename R>
    struct GlobalReduce
    {
        static __device__ void run(R& mymin, R& mymax, R* minval, R* maxval, int tid, int bid, R* sminval, R* smaxval)
        {
        #if __CUDA_ARCH__ >= 200
            if (tid == 0)
            {
                Emulation::glob::atomicMin(minval, mymin);
                Emulation::glob::atomicMax(maxval, mymax);
            }
        #else
            __shared__ bool is_last;

            if (tid == 0)
            {
                minval[bid] = mymin;
                maxval[bid] = mymax;

                __threadfence();

                unsigned int ticket = ::atomicAdd(&blocks_finished, 1);
                is_last = (ticket == gridDim.x * gridDim.y - 1);
            }

            __syncthreads();

            if (is_last)
            {
                int idx = ::min(tid, gridDim.x * gridDim.y - 1);

                mymin = minval[idx];
                mymax = maxval[idx];

                const minimum<R> minOp;
                const maximum<R> maxOp;
                device::reduce<BLOCK_SIZE>(smem_tuple(sminval, smaxval), thrust::tie(mymin, mymax), tid, thrust::make_tuple(minOp, maxOp));

                if (tid == 0)
                {
                    minval[0] = mymin;
                    maxval[0] = mymax;

                    blocks_finished = 0;
                }
            }
        #endif
        }
    };

    template <int BLOCK_SIZE, typename T, typename R, class Mask>
    __global__ void kernel(const PtrStepSz<T> src, const Mask mask, R* minval, R* maxval, const int twidth, const int theight)
    {
        __shared__ R sminval[BLOCK_SIZE];
        __shared__ R smaxval[BLOCK_SIZE];

        const int x0 = blockIdx.x * blockDim.x * twidth + threadIdx.x;
        const int y0 = blockIdx.y * blockDim.y * theight + threadIdx.y;

        const int tid = threadIdx.y * blockDim.x + threadIdx.x;
        const int bid = blockIdx.y * gridDim.x + blockIdx.x;

        R mymin = numeric_limits<R>::max();
        R mymax = -numeric_limits<R>::max();

        const minimum<R> minOp;
        const maximum<R> maxOp;

        for (int i = 0, y = y0; i < theight && y < src.rows; ++i, y += blockDim.y)
        {
            const T* ptr = src.ptr(y);

            for (int j = 0, x = x0; j < twidth && x < src.cols; ++j, x += blockDim.x)
            {
                if (mask(y, x))
                {
                    const R srcVal = ptr[x];

                    mymin = minOp(mymin, srcVal);
                    mymax = maxOp(mymax, srcVal);
                }
            }
        }

        device::reduce<BLOCK_SIZE>(smem_tuple(sminval, smaxval), thrust::tie(mymin, mymax), tid, thrust::make_tuple(minOp, maxOp));

        GlobalReduce<BLOCK_SIZE, R>::run(mymin, mymax, minval, maxval, tid, bid, sminval, smaxval);
    }

    const int threads_x = 32;
    const int threads_y = 8;

    void getLaunchCfg(int cols, int rows, dim3& block, dim3& grid)
    {
        block = dim3(threads_x, threads_y);

        grid = dim3(divUp(cols, block.x * block.y),
                    divUp(rows, block.y * block.x));

        grid.x = ::min(grid.x, block.x);
        grid.y = ::min(grid.y, block.y);
    }

    void getBufSize(int cols, int rows, int& bufcols, int& bufrows)
    {
        dim3 block, grid;
        getLaunchCfg(cols, rows, block, grid);

        bufcols = grid.x * grid.y * sizeof(double);
        bufrows = 2;
    }

    __global__ void setDefaultKernel(int* minval_buf, int* maxval_buf)
    {
        *minval_buf = numeric_limits<int>::max();
        *maxval_buf = numeric_limits<int>::min();
    }
    __global__ void setDefaultKernel(float* minval_buf, float* maxval_buf)
    {
        *minval_buf = numeric_limits<float>::max();
        *maxval_buf = -numeric_limits<float>::max();
    }
    __global__ void setDefaultKernel(double* minval_buf, double* maxval_buf)
    {
        *minval_buf = numeric_limits<double>::max();
        *maxval_buf = -numeric_limits<double>::max();
    }

    template <typename R>
    void setDefault(R* minval_buf, R* maxval_buf)
    {
        setDefaultKernel<<<1, 1>>>(minval_buf, maxval_buf);
    }

    template <typename T>
    void run(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf)
    {
        typedef typename MinMaxTypeTraits<T>::best_type R;

        dim3 block, grid;
        getLaunchCfg(src.cols, src.rows, block, grid);

        const int twidth = divUp(divUp(src.cols, grid.x), block.x);
        const int theight = divUp(divUp(src.rows, grid.y), block.y);

        R* minval_buf = (R*) buf.ptr(0);
        R* maxval_buf = (R*) buf.ptr(1);

        setDefault(minval_buf, maxval_buf);

        if (mask.data)
            kernel<threads_x * threads_y><<<grid, block>>>((PtrStepSz<T>) src, SingleMask(mask), minval_buf, maxval_buf, twidth, theight);
        else
            kernel<threads_x * threads_y><<<grid, block>>>((PtrStepSz<T>) src, WithOutMask(), minval_buf, maxval_buf, twidth, theight);

        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );

        R minval_, maxval_;
        cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(R), hipMemcpyDeviceToHost) );
        cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(R), hipMemcpyDeviceToHost) );
        *minval = minval_;
        *maxval = maxval_;
    }

    template void run<uchar >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<schar >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<ushort>(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<short >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<int   >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<float >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
    template void run<double>(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, PtrStepb buf);
}

#endif // CUDA_DISABLER
