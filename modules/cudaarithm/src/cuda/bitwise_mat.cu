/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace cv { namespace cuda { namespace device
{
    template <typename T> struct TransformFunctorTraits< bit_not<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_and<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_or<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< bit_xor<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <typename T> void bitMatNot(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, bit_not<T>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src, (PtrStepSz<T>) dst, bit_not<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatAnd(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_and<T>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_and<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatOr(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_or<T>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_or<T>(), WithOutMask(), stream);
    }

    template <typename T> void bitMatXor(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_xor<T>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, bit_xor<T>(), WithOutMask(), stream);
    }

    template void bitMatNot<uchar>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatNot<ushort>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatNot<uint>(PtrStepSzb src, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void bitMatAnd<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatAnd<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatAnd<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void bitMatOr<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatOr<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatOr<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void bitMatXor<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatXor<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void bitMatXor<uint>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
}

#endif // CUDA_DISABLER
