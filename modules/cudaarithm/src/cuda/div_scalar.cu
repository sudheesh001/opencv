/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <typename T, typename S, typename D> struct DivScalar : unary_function<T, D>
    {
        S val;

        __host__ explicit DivScalar(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a / val);
        }
    };

    template <typename T, typename S, typename D> struct DivScalarInv : unary_function<T, D>
    {
        S val;

        explicit DivScalarInv(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return a != 0 ? saturate_cast<D>(val / a) : 0;
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::DivScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };

    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::DivScalarInv<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void divScalar(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream)
    {
        if (inv)
        {
            DivScalarInv<T, S, D> op(static_cast<S>(val));
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
        else
        {
            DivScalar<T, S, D> op(static_cast<S>(val));
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
    }

    template void divScalar<uchar, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<uchar, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    template void divScalar<schar, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<schar, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<ushort, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<ushort, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<ushort, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<short, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<short, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<short, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<int, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<int, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<int, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<float, float, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<float, float, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<float, float, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<float, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);

    //template void divScalar<double, double, uchar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, schar>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, ushort>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, short>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, int>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    //template void divScalar<double, double, float>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
    template void divScalar<double, double, double>(PtrStepSzb src1, double val, bool inv, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
