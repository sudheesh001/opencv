#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct VAbsDiff4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff4(a, b);
        }

        __host__ __device__ __forceinline__ VAbsDiff4() {}
        __host__ __device__ __forceinline__ VAbsDiff4(const VAbsDiff4&) {}
    };

    struct VAbsDiff2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff2(a, b);
        }

        __host__ __device__ __forceinline__ VAbsDiff2() {}
        __host__ __device__ __forceinline__ VAbsDiff2(const VAbsDiff2&) {}
    };

    __device__ __forceinline__ int _abs(int a)
    {
        return ::abs(a);
    }
    __device__ __forceinline__ float _abs(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ double _abs(double a)
    {
        return ::fabs(a);
    }

    template <typename T> struct AbsDiffMat : binary_function<T, T, T>
    {
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return saturate_cast<T>(_abs(a - b));
        }

        __host__ __device__ __forceinline__ AbsDiffMat() {}
        __host__ __device__ __forceinline__ AbsDiffMat(const AbsDiffMat&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VAbsDiff4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <> struct TransformFunctorTraits< arithm::VAbsDiff2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T> struct TransformFunctorTraits< arithm::AbsDiffMat<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void absDiffMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VAbsDiff4(), WithOutMask(), stream);
    }

    void absDiffMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VAbsDiff2(), WithOutMask(), stream);
    }

    template <typename T>
    void absDiffMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, AbsDiffMat<T>(), WithOutMask(), stream);
    }

    template void absDiffMat<uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void absDiffMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
