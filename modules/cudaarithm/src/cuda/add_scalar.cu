/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <typename T, typename S, typename D> struct AddScalar : unary_function<T, D>
    {
        S val;

        __host__ explicit AddScalar(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a + val);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::AddScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void addScalar(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        AddScalar<T, S, D> op(static_cast<S>(val));

        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void addScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void addScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
}

#endif // CUDA_DISABLER
