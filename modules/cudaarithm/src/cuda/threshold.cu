/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace cv { namespace cuda { namespace device
{
    template <typename T> struct TransformFunctorTraits< thresh_binary_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_binary_inv_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_trunc_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_to_zero_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< thresh_to_zero_inv_func<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    template <template <typename> class Op, typename T>
    void threshold_caller(PtrStepSz<T> src, PtrStepSz<T> dst, T thresh, T maxVal, hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        device::transform(src, dst, op, WithOutMask(), stream);
    }

    template <typename T>
    void threshold(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream)
    {
        typedef void (*caller_t)(PtrStepSz<T> src, PtrStepSz<T> dst, T thresh, T maxVal, hipStream_t stream);

        static const caller_t callers[] =
        {
            threshold_caller<thresh_binary_func, T>,
            threshold_caller<thresh_binary_inv_func, T>,
            threshold_caller<thresh_trunc_func, T>,
            threshold_caller<thresh_to_zero_func, T>,
            threshold_caller<thresh_to_zero_inv_func, T>
        };

        callers[type]((PtrStepSz<T>) src, (PtrStepSz<T>) dst, static_cast<T>(thresh), static_cast<T>(maxVal), stream);
    }

    template void threshold<uchar>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<schar>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<ushort>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<short>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<int>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<float>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
    template void threshold<double>(PtrStepSzb src, PtrStepSzb dst, double thresh, double maxVal, int type, hipStream_t stream);
}

#endif // CUDA_DISABLER
