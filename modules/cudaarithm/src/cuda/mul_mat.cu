#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct Mul_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
            res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
            res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
            res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);

            return res;
        }

        __host__ __device__ __forceinline__ Mul_8uc4_32f() {}
        __host__ __device__ __forceinline__ Mul_8uc4_32f(const Mul_8uc4_32f&) {}
    };

    struct Mul_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return make_short4(saturate_cast<short>(a.x * b), saturate_cast<short>(a.y * b),
                               saturate_cast<short>(a.z * b), saturate_cast<short>(a.w * b));
        }

        __host__ __device__ __forceinline__ Mul_16sc4_32f() {}
        __host__ __device__ __forceinline__ Mul_16sc4_32f(const Mul_16sc4_32f&) {}
    };

    template <typename T, typename D> struct Mul : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a * b);
        }

        __host__ __device__ __forceinline__ Mul() {}
        __host__ __device__ __forceinline__ Mul(const Mul&) {}
    };

    template <typename T, typename S, typename D> struct MulScale : binary_function<T, T, D>
    {
        S scale;

        __host__ explicit MulScale(S scale_) : scale(scale_) {}

        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(scale * a * b);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<arithm::Mul_8uc4_32f> : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::Mul<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };

    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::MulScale<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void mulMat_8uc4_32f(PtrStepSz<uint> src1, PtrStepSzf src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, Mul_8uc4_32f(), WithOutMask(), stream);
    }

    void mulMat_16sc4_32f(PtrStepSz<short4> src1, PtrStepSzf src2, PtrStepSz<short4> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, Mul_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename S, typename D>
    void mulMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream)
    {
        if (scale == 1)
        {
            Mul<T, D> op;
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
        else
        {
            MulScale<T, S, D> op(static_cast<S>(scale));
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
    }

    template void mulMat<uchar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<uchar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    template void mulMat<schar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<schar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<ushort, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<ushort, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<ushort, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<short, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<short, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<short, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<int, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<int, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<int, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<float, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<float, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<float, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<float, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void mulMat<double, double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void mulMat<double, double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void mulMat<double, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
}

#endif // CUDA_DISABLER
