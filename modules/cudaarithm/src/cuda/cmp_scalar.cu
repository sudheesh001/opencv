/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"
#include "opencv2/core/cuda/vec_math.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <class Op, typename T>
    struct Cmp : binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T a, T b) const
        {
            Op op;
            return -op(a, b);
        }
    };

#define TYPE_VEC(type, cn) typename TypeVec<type, cn>::vec_type

    template <class Op, typename T, int cn> struct CmpScalar;
    template <class Op, typename T>
    struct CmpScalar<Op, T, 1> : unary_function<T, uchar>
    {
        T val;

        __host__ explicit CmpScalar(T val_) : val(val_) {}

        __device__ __forceinline__ uchar operator()(T src) const
        {
            Cmp<Op, T> op;
            return op(src, val);
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 2> : unary_function<TYPE_VEC(T, 2), TYPE_VEC(uchar, 2)>
    {
        TYPE_VEC(T, 2) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 2) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 2) operator()(const TYPE_VEC(T, 2) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 2)>::make(op(src.x, val.x), op(src.y, val.y));
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 3> : unary_function<TYPE_VEC(T, 3), TYPE_VEC(uchar, 3)>
    {
        TYPE_VEC(T, 3) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 3) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 3) operator()(const TYPE_VEC(T, 3) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 3)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z));
        }
    };
    template <class Op, typename T>
    struct CmpScalar<Op, T, 4> : unary_function<TYPE_VEC(T, 4), TYPE_VEC(uchar, 4)>
    {
        TYPE_VEC(T, 4) val;

        __host__ explicit CmpScalar(TYPE_VEC(T, 4) val_) : val(val_) {}

        __device__ __forceinline__ TYPE_VEC(uchar, 4) operator()(const TYPE_VEC(T, 4) & src) const
        {
            Cmp<Op, T> op;
            return VecTraits<TYPE_VEC(uchar, 4)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z), op(src.w, val.w));
        }
    };

#undef TYPE_VEC
}

namespace cv { namespace cuda { namespace device
{
    template <class Op, typename T> struct TransformFunctorTraits< arithm::CmpScalar<Op, T, 1> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(uchar)>
    {
    };
}}}

namespace arithm
{
    template <template <typename> class Op, typename T, int cn>
    void cmpScalar(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef typename TypeVec<T, cn>::vec_type src_t;
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        T sval[] = {static_cast<T>(val[0]), static_cast<T>(val[1]), static_cast<T>(val[2]), static_cast<T>(val[3])};
        src_t val1 = VecTraits<src_t>::make(sval);

        CmpScalar<Op<T>, T, cn> op(val1);
        device::transform((PtrStepSz<src_t>) src, (PtrStepSz<dst_t>) dst, op, WithOutMask(), stream);
    }

    template <typename T> void cmpScalarEq(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<equal_to, T, 1>,
            cmpScalar<equal_to, T, 2>,
            cmpScalar<equal_to, T, 3>,
            cmpScalar<equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarNe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<not_equal_to, T, 1>,
            cmpScalar<not_equal_to, T, 2>,
            cmpScalar<not_equal_to, T, 3>,
            cmpScalar<not_equal_to, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarLt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<less, T, 1>,
            cmpScalar<less, T, 2>,
            cmpScalar<less, T, 3>,
            cmpScalar<less, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarLe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<less_equal, T, 1>,
            cmpScalar<less_equal, T, 2>,
            cmpScalar<less_equal, T, 3>,
            cmpScalar<less_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarGt(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<greater, T, 1>,
            cmpScalar<greater, T, 2>,
            cmpScalar<greater, T, 3>,
            cmpScalar<greater, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }
    template <typename T> void cmpScalarGe(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, double val[4], PtrStepSzb dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            0,
            cmpScalar<greater_equal, T, 1>,
            cmpScalar<greater_equal, T, 2>,
            cmpScalar<greater_equal, T, 3>,
            cmpScalar<greater_equal, T, 4>
        };

        funcs[cn](src, val, dst, stream);
    }

    template void cmpScalarEq<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarEq<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void cmpScalarNe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarNe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void cmpScalarLt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void cmpScalarLe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarLe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void cmpScalarGt<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGt<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);

    template void cmpScalarGe<uchar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<schar >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<ushort>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<short >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<int   >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<float >(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
    template void cmpScalarGe<double>(PtrStepSzb src, int cn, double val[4], PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
