/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct VCmpEq4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmpeq4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpEq4() {}
        __host__ __device__ __forceinline__ VCmpEq4(const VCmpEq4&) {}
    };
    struct VCmpNe4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmpne4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpNe4() {}
        __host__ __device__ __forceinline__ VCmpNe4(const VCmpNe4&) {}
    };
    struct VCmpLt4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmplt4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpLt4() {}
        __host__ __device__ __forceinline__ VCmpLt4(const VCmpLt4&) {}
    };
    struct VCmpLe4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vcmple4(a, b);
        }

        __host__ __device__ __forceinline__ VCmpLe4() {}
        __host__ __device__ __forceinline__ VCmpLe4(const VCmpLe4&) {}
    };

    template <class Op, typename T>
    struct Cmp : binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T a, T b) const
        {
            Op op;
            return -op(a, b);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VCmpEq4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpNe4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpLt4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };
    template <> struct TransformFunctorTraits< arithm::VCmpLe4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <class Op, typename T> struct TransformFunctorTraits< arithm::Cmp<Op, T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(uchar)>
    {
    };
}}}

namespace arithm
{
    void cmpMatEq_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VCmpEq4(), WithOutMask(), stream);
    }
    void cmpMatNe_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VCmpNe4(), WithOutMask(), stream);
    }
    void cmpMatLt_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VCmpLt4(), WithOutMask(), stream);
    }
    void cmpMatLe_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VCmpLe4(), WithOutMask(), stream);
    }

    template <template <typename> class Op, typename T>
    void cmpMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        Cmp<Op<T>, T> op;
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, dst, op, WithOutMask(), stream);
    }

    template <typename T> void cmpMatEq(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatNe(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<not_equal_to, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatLt(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<less, T>(src1, src2, dst, stream);
    }
    template <typename T> void cmpMatLe(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        cmpMat<less_equal, T>(src1, src2, dst, stream);
    }

    template void cmpMatEq<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatEq<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void cmpMatNe<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatNe<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void cmpMatLt<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLt<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template void cmpMatLe<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void cmpMatLe<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
