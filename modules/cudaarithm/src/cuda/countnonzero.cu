#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/emulation.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace countNonZero
{
    __device__ unsigned int blocks_finished = 0;

    template <int BLOCK_SIZE, typename T>
    __global__ void kernel(const PtrStepSz<T> src, unsigned int* count, const int twidth, const int theight)
    {
        __shared__ unsigned int scount[BLOCK_SIZE];

        const int x0 = blockIdx.x * blockDim.x * twidth + threadIdx.x;
        const int y0 = blockIdx.y * blockDim.y * theight + threadIdx.y;

        const int tid = threadIdx.y * blockDim.x + threadIdx.x;

        unsigned int mycount = 0;

        for (int i = 0, y = y0; i < theight && y < src.rows; ++i, y += blockDim.y)
        {
            const T* ptr = src.ptr(y);

            for (int j = 0, x = x0; j < twidth && x < src.cols; ++j, x += blockDim.x)
            {
                const T srcVal = ptr[x];

                mycount += (srcVal != 0);
            }
        }

        device::reduce<BLOCK_SIZE>(scount, mycount, tid, plus<unsigned int>());

    #if __CUDA_ARCH__ >= 200
        if (tid == 0)
            ::atomicAdd(count, mycount);
    #else
        __shared__ bool is_last;
        const int bid = blockIdx.y * gridDim.x + blockIdx.x;

        if (tid == 0)
        {
            count[bid] = mycount;

            __threadfence();

            unsigned int ticket = ::atomicInc(&blocks_finished, gridDim.x * gridDim.y);
            is_last = (ticket == gridDim.x * gridDim.y - 1);
        }

        __syncthreads();

        if (is_last)
        {
            mycount = tid < gridDim.x * gridDim.y ? count[tid] : 0;

            device::reduce<BLOCK_SIZE>(scount, mycount, tid, plus<unsigned int>());

            if (tid == 0)
            {
                count[0] = mycount;

                blocks_finished = 0;
            }
        }
    #endif
    }

    const int threads_x = 32;
    const int threads_y = 8;

    void getLaunchCfg(int cols, int rows, dim3& block, dim3& grid)
    {
        block = dim3(threads_x, threads_y);

        grid = dim3(divUp(cols, block.x * block.y),
                    divUp(rows, block.y * block.x));

        grid.x = ::min(grid.x, block.x);
        grid.y = ::min(grid.y, block.y);
    }

    void getBufSize(int cols, int rows, int& bufcols, int& bufrows)
    {
        dim3 block, grid;
        getLaunchCfg(cols, rows, block, grid);

        bufcols = grid.x * grid.y * sizeof(int);
        bufrows = 1;
    }

    template <typename T>
    int run(const PtrStepSzb src, PtrStep<unsigned int> buf)
    {
        dim3 block, grid;
        getLaunchCfg(src.cols, src.rows, block, grid);

        const int twidth = divUp(divUp(src.cols, grid.x), block.x);
        const int theight = divUp(divUp(src.rows, grid.y), block.y);

        unsigned int* count_buf = buf.ptr(0);

        cudaSafeCall( hipMemset(count_buf, 0, sizeof(unsigned int)) );

        kernel<threads_x * threads_y><<<grid, block>>>((PtrStepSz<T>) src, count_buf, twidth, theight);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );

        unsigned int count;
        cudaSafeCall(hipMemcpy(&count, count_buf, sizeof(unsigned int), hipMemcpyDeviceToHost));

        return count;
    }

    template int run<uchar >(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<schar >(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<ushort>(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<short >(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<int   >(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<float >(const PtrStepSzb src, PtrStep<unsigned int> buf);
    template int run<double>(const PtrStepSzb src, PtrStep<unsigned int> buf);
}

#endif // CUDA_DISABLER
