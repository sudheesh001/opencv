#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/utility.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace minMaxLoc
{
    // To avoid shared bank conflicts we convert each value into value of
    // appropriate type (32 bits minimum)
    template <typename T> struct MinMaxTypeTraits;
    template <> struct MinMaxTypeTraits<unsigned char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<signed char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<unsigned short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<int> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<float> { typedef float best_type; };
    template <> struct MinMaxTypeTraits<double> { typedef double best_type; };

    template <int BLOCK_SIZE, typename T, class Mask>
    __global__ void kernel_pass_1(const PtrStepSz<T> src, const Mask mask, T* minval, T* maxval, unsigned int* minloc, unsigned int* maxloc, const int twidth, const int theight)
    {
        typedef typename MinMaxTypeTraits<T>::best_type work_type;

        __shared__ work_type sminval[BLOCK_SIZE];
        __shared__ work_type smaxval[BLOCK_SIZE];
        __shared__ unsigned int sminloc[BLOCK_SIZE];
        __shared__ unsigned int smaxloc[BLOCK_SIZE];

        const int x0 = blockIdx.x * blockDim.x * twidth + threadIdx.x;
        const int y0 = blockIdx.y * blockDim.y * theight + threadIdx.y;

        const int tid = threadIdx.y * blockDim.x + threadIdx.x;
        const int bid = blockIdx.y * gridDim.x + blockIdx.x;

        work_type mymin = numeric_limits<work_type>::max();
        work_type mymax = -numeric_limits<work_type>::max();
        unsigned int myminloc = 0;
        unsigned int mymaxloc = 0;

        for (int i = 0, y = y0; i < theight && y < src.rows; ++i, y += blockDim.y)
        {
            const T* ptr = src.ptr(y);

            for (int j = 0, x = x0; j < twidth && x < src.cols; ++j, x += blockDim.x)
            {
                if (mask(y, x))
                {
                    const work_type srcVal = ptr[x];

                    if (srcVal < mymin)
                    {
                        mymin = srcVal;
                        myminloc = y * src.cols + x;
                    }

                    if (srcVal > mymax)
                    {
                        mymax = srcVal;
                        mymaxloc = y * src.cols + x;
                    }
                }
            }
        }

        reduceKeyVal<BLOCK_SIZE>(smem_tuple(sminval, smaxval), thrust::tie(mymin, mymax),
                                 smem_tuple(sminloc, smaxloc), thrust::tie(myminloc, mymaxloc),
                                 tid,
                                 thrust::make_tuple(less<work_type>(), greater<work_type>()));

        if (tid == 0)
        {
            minval[bid] = (T) mymin;
            maxval[bid] = (T) mymax;
            minloc[bid] = myminloc;
            maxloc[bid] = mymaxloc;
        }
    }
    template <int BLOCK_SIZE, typename T>
    __global__ void kernel_pass_2(T* minval, T* maxval, unsigned int* minloc, unsigned int* maxloc, int count)
    {
        typedef typename MinMaxTypeTraits<T>::best_type work_type;

        __shared__ work_type sminval[BLOCK_SIZE];
        __shared__ work_type smaxval[BLOCK_SIZE];
        __shared__ unsigned int sminloc[BLOCK_SIZE];
        __shared__ unsigned int smaxloc[BLOCK_SIZE];

        unsigned int idx = ::min(threadIdx.x, count - 1);

        work_type mymin = minval[idx];
        work_type mymax = maxval[idx];
        unsigned int myminloc = minloc[idx];
        unsigned int mymaxloc = maxloc[idx];

        reduceKeyVal<BLOCK_SIZE>(smem_tuple(sminval, smaxval), thrust::tie(mymin, mymax),
                                 smem_tuple(sminloc, smaxloc), thrust::tie(myminloc, mymaxloc),
                                 threadIdx.x,
                                 thrust::make_tuple(less<work_type>(), greater<work_type>()));

        if (threadIdx.x == 0)
        {
            minval[0] = (T) mymin;
            maxval[0] = (T) mymax;
            minloc[0] = myminloc;
            maxloc[0] = mymaxloc;
        }
    }

    const int threads_x = 32;
    const int threads_y = 8;

    void getLaunchCfg(int cols, int rows, dim3& block, dim3& grid)
    {
        block = dim3(threads_x, threads_y);

        grid = dim3(divUp(cols, block.x * block.y),
                    divUp(rows, block.y * block.x));

        grid.x = ::min(grid.x, block.x);
        grid.y = ::min(grid.y, block.y);
    }

    void getBufSize(int cols, int rows, size_t elem_size, int& b1cols, int& b1rows, int& b2cols, int& b2rows)
    {
        dim3 block, grid;
        getLaunchCfg(cols, rows, block, grid);

        // For values
        b1cols = (int)(grid.x * grid.y * elem_size);
        b1rows = 2;

        // For locations
        b2cols = grid.x * grid.y * sizeof(int);
        b2rows = 2;
    }

    template <typename T>
    void run(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf)
    {
        dim3 block, grid;
        getLaunchCfg(src.cols, src.rows, block, grid);

        const int twidth = divUp(divUp(src.cols, grid.x), block.x);
        const int theight = divUp(divUp(src.rows, grid.y), block.y);

        T* minval_buf = (T*) valbuf.ptr(0);
        T* maxval_buf = (T*) valbuf.ptr(1);
        unsigned int* minloc_buf = locbuf.ptr(0);
        unsigned int* maxloc_buf = locbuf.ptr(1);

        if (mask.data)
            kernel_pass_1<threads_x * threads_y><<<grid, block>>>((PtrStepSz<T>) src, SingleMask(mask), minval_buf, maxval_buf, minloc_buf, maxloc_buf, twidth, theight);
        else
            kernel_pass_1<threads_x * threads_y><<<grid, block>>>((PtrStepSz<T>) src, WithOutMask(), minval_buf, maxval_buf, minloc_buf, maxloc_buf, twidth, theight);

        cudaSafeCall( hipGetLastError() );

        kernel_pass_2<threads_x * threads_y><<<1, threads_x * threads_y>>>(minval_buf, maxval_buf, minloc_buf, maxloc_buf, grid.x * grid.y);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );

        T minval_, maxval_;
        cudaSafeCall( hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost) );
        cudaSafeCall( hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost) );
        *minval = minval_;
        *maxval = maxval_;

        unsigned int minloc_, maxloc_;
        cudaSafeCall( hipMemcpy(&minloc_, minloc_buf, sizeof(unsigned int), hipMemcpyDeviceToHost) );
        cudaSafeCall( hipMemcpy(&maxloc_, maxloc_buf, sizeof(unsigned int), hipMemcpyDeviceToHost) );
        minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
        maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
    }

    template void run<unsigned char >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<signed char >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<unsigned short>(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<short >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<int   >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<float >(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
    template void run<double>(const PtrStepSzb src, const PtrStepb mask, double* minval, double* maxval, int* minloc, int* maxloc, PtrStepb valbuf, PtrStep<unsigned int> locbuf);
}

#endif // CUDA_DISABLER
