/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    template <typename T, typename S, typename D> struct MulScalar : unary_function<T, D>
    {
        S val;

        __host__ explicit MulScalar(S val_) : val(val_) {}

        __device__ __forceinline__ D operator ()(T a) const
        {
            return saturate_cast<D>(a * val);
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::MulScalar<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    template <typename T, typename S, typename D>
    void mulScalar(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream)
    {
        MulScalar<T, S, D> op(static_cast<S>(val));
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
    }

    template void mulScalar<uchar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<uchar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    template void mulScalar<schar, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<schar, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<ushort, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<ushort, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<ushort, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<short, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<short, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<short, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<int, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<int, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<int, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<float, float, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<float, float, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<float, float, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<float, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);

    //template void mulScalar<double, double, uchar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, schar>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, ushort>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, short>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, int>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    //template void mulScalar<double, double, float>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
    template void mulScalar<double, double, double>(PtrStepSzb src1, double val, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
