#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct Div_8uc4_32f : binary_function<uint, float, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, float b) const
        {
            uint res = 0;

            if (b != 0)
            {
                b = 1.0f / b;
                res |= (saturate_cast<uchar>((0xffu & (a      )) * b)      );
                res |= (saturate_cast<uchar>((0xffu & (a >>  8)) * b) <<  8);
                res |= (saturate_cast<uchar>((0xffu & (a >> 16)) * b) << 16);
                res |= (saturate_cast<uchar>((0xffu & (a >> 24)) * b) << 24);
            }

            return res;
        }
    };

    struct Div_16sc4_32f : binary_function<short4, float, short4>
    {
        __device__ __forceinline__ short4 operator ()(short4 a, float b) const
        {
            return b != 0 ? make_short4(saturate_cast<short>(a.x / b), saturate_cast<short>(a.y / b),
                                        saturate_cast<short>(a.z / b), saturate_cast<short>(a.w / b))
                          : make_short4(0,0,0,0);
        }
    };

    template <typename T, typename D> struct Div : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(a / b) : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };
    template <typename T> struct Div<T, float> : binary_function<T, T, float>
    {
        __device__ __forceinline__ float operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<float>(a) / b : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };
    template <typename T> struct Div<T, double> : binary_function<T, T, double>
    {
        __device__ __forceinline__ double operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<double>(a) / b : 0;
        }

        __host__ __device__ __forceinline__ Div() {}
        __host__ __device__ __forceinline__ Div(const Div&) {}
    };

    template <typename T, typename S, typename D> struct DivScale : binary_function<T, T, D>
    {
        S scale;

        __host__ explicit DivScale(S scale_) : scale(scale_) {}

        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(scale * a / b) : 0;
        }
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<arithm::Div_8uc4_32f> : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::Div<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };

    template <typename T, typename S, typename D> struct TransformFunctorTraits< arithm::DivScale<T, S, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void divMat_8uc4_32f(PtrStepSz<uint> src1, PtrStepSzf src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, Div_8uc4_32f(), WithOutMask(), stream);
    }

    void divMat_16sc4_32f(PtrStepSz<short4> src1, PtrStepSzf src2, PtrStepSz<short4> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, Div_16sc4_32f(), WithOutMask(), stream);
    }

    template <typename T, typename S, typename D>
    void divMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream)
    {
        if (scale == 1)
        {
            Div<T, D> op;
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
        else
        {
            DivScale<T, S, D> op(static_cast<S>(scale));
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, op, WithOutMask(), stream);
        }
    }

    template void divMat<uchar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<uchar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    template void divMat<schar, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<schar, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<ushort, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<ushort, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<ushort, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<short, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<short, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<short, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<int, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<int, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<int, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<float, float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<float, float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<float, float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<float, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);

    //template void divMat<double, double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    //template void divMat<double, double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
    template void divMat<double, double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, double scale, hipStream_t stream);
}

#endif // CUDA_DISABLER
