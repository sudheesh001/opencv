/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace arithm
{
    struct VAdd4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vadd4(a, b);
        }

        __host__ __device__ __forceinline__ VAdd4() {}
        __host__ __device__ __forceinline__ VAdd4(const VAdd4&) {}
    };

    struct VAdd2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vadd2(a, b);
        }

        __host__ __device__ __forceinline__ VAdd2() {}
        __host__ __device__ __forceinline__ VAdd2(const VAdd2&) {}
    };

    template <typename T, typename D> struct AddMat : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a + b);
        }

        __host__ __device__ __forceinline__ AddMat() {}
        __host__ __device__ __forceinline__ AddMat(const AddMat&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VAdd4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <> struct TransformFunctorTraits< arithm::VAdd2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T, typename D> struct TransformFunctorTraits< arithm::AddMat<T, D> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(D)>
    {
    };
}}}

namespace arithm
{
    void addMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VAdd4(), WithOutMask(), stream);
    }

    void addMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VAdd2(), WithOutMask(), stream);
    }

    template <typename T, typename D>
    void addMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream)
    {
        if (mask.data)
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, AddMat<T, D>(), mask, stream);
        else
            device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<D>) dst, AddMat<T, D>(), WithOutMask(), stream);
    }

    template void addMat<uchar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<uchar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    template void addMat<schar, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<schar, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<ushort, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<ushort, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<ushort, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<short, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<short, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<short, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<int, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<int, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<int, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<float, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<float, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<float, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<float, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);

    //template void addMat<double, uchar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, schar>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, short>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, int>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    //template void addMat<double, float>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
    template void addMat<double, double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, PtrStepb mask, hipStream_t stream);
}

#endif // CUDA_DISABLER
