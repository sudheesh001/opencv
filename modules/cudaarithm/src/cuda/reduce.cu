#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/limits.hpp"

#include "unroll_detail.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace reduce
{
    struct Sum
    {
        template <typename T>
        __device__ __forceinline__ T startValue() const
        {
            return VecTraits<T>::all(0);
        }

        template <typename T>
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return a + b;
        }

        template <typename T>
        __device__ __forceinline__ T result(T r, int) const
        {
            return r;
        }

        __host__ __device__ __forceinline__ Sum() {}
        __host__ __device__ __forceinline__ Sum(const Sum&) {}
    };

    template <typename T> struct OutputType
    {
        typedef float type;
    };
    template <> struct OutputType<double>
    {
        typedef double type;
    };

    struct Avg
    {
        template <typename T>
        __device__ __forceinline__ T startValue() const
        {
            return VecTraits<T>::all(0);
        }

        template <typename T>
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return a + b;
        }

        template <typename T>
        __device__ __forceinline__ typename TypeVec<typename OutputType<typename VecTraits<T>::elem_type>::type, VecTraits<T>::cn>::vec_type result(T r, float sz) const
        {
            return r / sz;
        }

        __host__ __device__ __forceinline__ Avg() {}
        __host__ __device__ __forceinline__ Avg(const Avg&) {}
    };

    struct Min
    {
        template <typename T>
        __device__ __forceinline__ T startValue() const
        {
            return VecTraits<T>::all(numeric_limits<typename VecTraits<T>::elem_type>::max());
        }

        template <typename T>
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            minimum<T> minOp;
            return minOp(a, b);
        }

        template <typename T>
        __device__ __forceinline__ T result(T r, int) const
        {
            return r;
        }

        __host__ __device__ __forceinline__ Min() {}
        __host__ __device__ __forceinline__ Min(const Min&) {}
    };

    struct Max
    {
        template <typename T>
        __device__ __forceinline__ T startValue() const
        {
            return VecTraits<T>::all(-numeric_limits<typename VecTraits<T>::elem_type>::max());
        }

        template <typename T>
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            maximum<T> maxOp;
            return maxOp(a, b);
        }

        template <typename T>
        __device__ __forceinline__ T result(T r, int) const
        {
            return r;
        }

        __host__ __device__ __forceinline__ Max() {}
        __host__ __device__ __forceinline__ Max(const Max&) {}
    };

    ///////////////////////////////////////////////////////////

    template <typename T, typename S, typename D, class Op>
    __global__ void rowsKernel(const PtrStepSz<T> src, D* dst, const Op op)
    {
        __shared__ S smem[16 * 16];

        const int x = blockIdx.x * 16 + threadIdx.x;

        S myVal = op.template startValue<S>();

        if (x < src.cols)
        {
            for (int y = threadIdx.y; y < src.rows; y += 16)
            {
                S srcVal = src(y, x);
                myVal = op(myVal, srcVal);
            }
        }

        smem[threadIdx.x * 16 + threadIdx.y] = myVal;

        __syncthreads();

        volatile S* srow = smem + threadIdx.y * 16;

        myVal = srow[threadIdx.x];
        device::reduce<16>(srow, myVal, threadIdx.x, op);

        if (threadIdx.x == 0)
            srow[0] = myVal;

        __syncthreads();

        if (threadIdx.y == 0 && x < src.cols)
            dst[x] = (D) op.result(smem[threadIdx.x * 16], src.rows);
    }

    template <typename T, typename S, typename D, class Op>
    void rowsCaller(PtrStepSz<T> src, D* dst, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(src.cols, block.x));

        Op op;
        rowsKernel<T, S, D, Op><<<grid, block, 0, stream>>>(src, dst, op);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T, typename S, typename D>
    void rows(PtrStepSzb src, void* dst, int op, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSz<T> src, D* dst, hipStream_t stream);
        static const func_t funcs[] =
        {
            rowsCaller<T, S, D, Sum>,
            rowsCaller<T, S, D, Avg>,
            rowsCaller<T, S, D, Max>,
            rowsCaller<T, S, D, Min>
        };

        funcs[op]((PtrStepSz<T>) src, (D*) dst, stream);
    }

    template void rows<unsigned char, int, unsigned char>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned char, int, int>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned char, float, float>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned char, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    template void rows<unsigned short, int, unsigned short>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned short, int, int>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned short, float, float>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<unsigned short, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    template void rows<short, int, short>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<short, int, int>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<short, float, float>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<short, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    template void rows<int, int, int>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<int, float, float>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<int, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    template void rows<float, float, float>(PtrStepSzb src, void* dst, int op, hipStream_t stream);
    template void rows<float, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    template void rows<double, double, double>(PtrStepSzb src, void* dst, int op, hipStream_t stream);

    ///////////////////////////////////////////////////////////

    template <int BLOCK_SIZE, typename T, typename S, typename D, int cn, class Op>
    __global__ void colsKernel(const PtrStepSz<typename TypeVec<T, cn>::vec_type> src, typename TypeVec<D, cn>::vec_type* dst, const Op op)
    {
        typedef typename TypeVec<T, cn>::vec_type src_type;
        typedef typename TypeVec<S, cn>::vec_type work_type;
        typedef typename TypeVec<D, cn>::vec_type dst_type;

        __shared__ S smem[BLOCK_SIZE * cn];

        const int y = blockIdx.x;

        const src_type* srcRow = src.ptr(y);

        work_type myVal = op.template startValue<work_type>();

        for (int x = threadIdx.x; x < src.cols; x += BLOCK_SIZE)
            myVal = op(myVal, saturate_cast<work_type>(srcRow[x]));

        device::reduce<BLOCK_SIZE>(detail::Unroll<cn>::template smem_tuple<BLOCK_SIZE>(smem), detail::Unroll<cn>::tie(myVal), threadIdx.x, detail::Unroll<cn>::op(op));

        if (threadIdx.x == 0)
            dst[y] = saturate_cast<dst_type>(op.result(myVal, src.cols));
    }

    template <typename T, typename S, typename D, int cn, class Op> void colsCaller(PtrStepSzb src, void* dst, hipStream_t stream)
    {
        const int BLOCK_SIZE = 256;

        const dim3 block(BLOCK_SIZE);
        const dim3 grid(src.rows);

        Op op;
        colsKernel<BLOCK_SIZE, T, S, D, cn, Op><<<grid, block, 0, stream>>>((PtrStepSz<typename TypeVec<T, cn>::vec_type>) src, (typename TypeVec<D, cn>::vec_type*) dst, op);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );

    }

    template <typename T, typename S, typename D> void cols(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream)
    {
        typedef void (*func_t)(PtrStepSzb src, void* dst, hipStream_t stream);
        static const func_t funcs[5][4] =
        {
            {0,0,0,0},
            {colsCaller<T, S, D, 1, Sum>, colsCaller<T, S, D, 1, Avg>, colsCaller<T, S, D, 1, Max>, colsCaller<T, S, D, 1, Min>},
            {colsCaller<T, S, D, 2, Sum>, colsCaller<T, S, D, 2, Avg>, colsCaller<T, S, D, 2, Max>, colsCaller<T, S, D, 2, Min>},
            {colsCaller<T, S, D, 3, Sum>, colsCaller<T, S, D, 3, Avg>, colsCaller<T, S, D, 3, Max>, colsCaller<T, S, D, 3, Min>},
            {colsCaller<T, S, D, 4, Sum>, colsCaller<T, S, D, 4, Avg>, colsCaller<T, S, D, 4, Max>, colsCaller<T, S, D, 4, Min>},
        };

        funcs[cn][op](src, dst, stream);
    }

    template void cols<unsigned char, int, unsigned char>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned char, int, int>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned char, float, float>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned char, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);

    template void cols<unsigned short, int, unsigned short>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned short, int, int>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned short, float, float>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<unsigned short, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);

    template void cols<short, int, short>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<short, int, int>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<short, float, float>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<short, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);

    template void cols<int, int, int>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<int, float, float>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<int, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);

    template void cols<float, float, float>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
    template void cols<float, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);

    template void cols<double, double, double>(PtrStepSzb src, void* dst, int cn, int op, hipStream_t stream);
}

#endif /* CUDA_DISABLER */
