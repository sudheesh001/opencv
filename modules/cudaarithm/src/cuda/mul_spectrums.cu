#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "cvconfig.h"

#ifdef HAVE_CUFFT

#include <hipfft/hipfft.h>

#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace cuda { namespace device
{
    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums

    __global__ void mulSpectrumsKernel(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows)
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
        }
    }


    void mulSpectrums(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c, hipStream_t stream)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel<<<grid, threads, 0, stream>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulSpectrums_CONJ

    __global__ void mulSpectrumsKernel_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows)
        {
            c.ptr(y)[x] = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
        }
    }


    void mulSpectrums_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, PtrStepSz<hipfftComplex> c, hipStream_t stream)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulSpectrumsKernel_CONJ<<<grid, threads, 0, stream>>>(a, b, c);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums

    __global__ void mulAndScaleSpectrumsKernel(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows)
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], b.ptr(y)[x]);
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c, hipStream_t stream)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel<<<grid, threads, 0, stream>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        if (stream)
            cudaSafeCall( hipDeviceSynchronize() );
    }


    //////////////////////////////////////////////////////////////////////////
    // mulAndScaleSpectrums_CONJ

    __global__ void mulAndScaleSpectrumsKernel_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < c.cols && y < c.rows)
        {
            hipfftComplex v = hipCmulf(a.ptr(y)[x], hipConjf(b.ptr(y)[x]));
            c.ptr(y)[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
        }
    }


    void mulAndScaleSpectrums_CONJ(const PtrStep<hipfftComplex> a, const PtrStep<hipfftComplex> b, float scale, PtrStepSz<hipfftComplex> c, hipStream_t stream)
    {
        dim3 threads(256);
        dim3 grid(divUp(c.cols, threads.x), divUp(c.rows, threads.y));

        mulAndScaleSpectrumsKernel_CONJ<<<grid, threads, 0, stream>>>(a, b, scale, c);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}}} // namespace cv { namespace cuda { namespace cudev

#endif // HAVE_CUFFT

#endif /* CUDA_DISABLER */
