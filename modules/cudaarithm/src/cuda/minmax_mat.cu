#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/simd_functions.hpp"

#include "arithm_func_traits.hpp"

using namespace cv::cuda;
using namespace cv::cuda::device;

//////////////////////////////////////////////////////////////////////////
// min

namespace arithm
{
    struct VMin4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin4(a, b);
        }

        __host__ __device__ __forceinline__ VMin4() {}
        __host__ __device__ __forceinline__ VMin4(const VMin4&) {}
    };

    struct VMin2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin2(a, b);
        }

        __host__ __device__ __forceinline__ VMin2() {}
        __host__ __device__ __forceinline__ VMin2(const VMin2&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VMin4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <> struct TransformFunctorTraits< arithm::VMin2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T> struct TransformFunctorTraits< minimum<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< minimum<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void minMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VMin4(), WithOutMask(), stream);
    }

    void minMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VMin2(), WithOutMask(), stream);
    }

    template <typename T> void minMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, minimum<T>(), WithOutMask(), stream);
    }

    template void minMat<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void minMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template <typename T> void minScalar(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::cuda::device::bind2nd(minimum<T>(), src2), WithOutMask(), stream);
    }

    template void minScalar<uchar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<schar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<ushort>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<short >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<int   >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<float >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void minScalar<double>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
}

//////////////////////////////////////////////////////////////////////////
// max

namespace arithm
{
    struct VMax4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax4(a, b);
        }

        __host__ __device__ __forceinline__ VMax4() {}
        __host__ __device__ __forceinline__ VMax4(const VMax4&) {}
    };

    struct VMax2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax2(a, b);
        }

        __host__ __device__ __forceinline__ VMax2() {}
        __host__ __device__ __forceinline__ VMax2(const VMax2&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits< arithm::VMax4 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <> struct TransformFunctorTraits< arithm::VMax2 > : arithm::ArithmFuncTraits<sizeof(uint), sizeof(uint)>
    {
    };

    template <typename T> struct TransformFunctorTraits< maximum<T> > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };

    template <typename T> struct TransformFunctorTraits< binder2nd< maximum<T> > > : arithm::ArithmFuncTraits<sizeof(T), sizeof(T)>
    {
    };
}}}

namespace arithm
{
    void maxMat_v4(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VMax4(), WithOutMask(), stream);
    }

    void maxMat_v2(PtrStepSz<uint> src1, PtrStepSz<uint> src2, PtrStepSz<uint> dst, hipStream_t stream)
    {
        device::transform(src1, src2, dst, VMax2(), WithOutMask(), stream);
    }

    template <typename T> void maxMat(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) src2, (PtrStepSz<T>) dst, maximum<T>(), WithOutMask(), stream);
    }

    template void maxMat<uchar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<schar >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<ushort>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<short >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<int   >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<float >(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);
    template void maxMat<double>(PtrStepSzb src1, PtrStepSzb src2, PtrStepSzb dst, hipStream_t stream);

    template <typename T> void maxScalar(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream)
    {
        device::transform((PtrStepSz<T>) src1, (PtrStepSz<T>) dst, cv::cuda::device::bind2nd(maximum<T>(), src2), WithOutMask(), stream);
    }

    template void maxScalar<uchar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<schar >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<ushort>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<short >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<int   >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<float >(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
    template void maxScalar<double>(PtrStepSzb src1, double src2, PtrStepSzb dst, hipStream_t stream);
}

#endif // CUDA_DISABLER
