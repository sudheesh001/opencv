#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"

namespace cv { namespace cuda { namespace device
{
    __device__ __forceinline__ void rgb_to_y(const uchar b, const uchar g, const uchar r, uchar& y)
    {
        y = static_cast<uchar>(((int)(30 * r) + (int)(59 * g) + (int)(11 * b)) / 100);
    }

    __device__ __forceinline__ void rgb_to_yuv(const uchar b, const uchar g, const uchar r, uchar& y, uchar& u, uchar& v)
    {
        rgb_to_y(b, g, r, y);
        u = static_cast<uchar>(((int)(-17 * r) - (int)(33 * g) + (int)(50 * b) + 12800) / 100);
        v = static_cast<uchar>(((int)(50 * r) - (int)(42 * g) - (int)(8 * b) + 12800) / 100);
    }

    __global__ void Gray_to_YV12(const PtrStepSzb src, PtrStepb dst)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
        const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

        if (x + 1 >= src.cols || y + 1 >= src.rows)
            return;

        // get pointers to the data
        const size_t planeSize = src.rows * dst.step;
        PtrStepb y_plane(dst.data, dst.step);
        PtrStepb u_plane(y_plane.data + planeSize, dst.step / 2);
        PtrStepb v_plane(u_plane.data + (planeSize / 4), dst.step / 2);

        uchar pix;
        uchar y_val, u_val, v_val;

        pix = src(y, x);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y, x) = y_val;

        pix = src(y, x + 1);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y, x + 1) = y_val;

        pix = src(y + 1, x);
        rgb_to_y(pix, pix, pix, y_val);
        y_plane(y + 1, x) = y_val;

        pix = src(y + 1, x + 1);
        rgb_to_yuv(pix, pix, pix, y_val, u_val, v_val);
        y_plane(y + 1, x + 1) = y_val;
        u_plane(y / 2, x / 2) = u_val;
        v_plane(y / 2, x / 2) = v_val;
    }

    template <typename T>
    __global__ void RGB_to_YV12(const PtrStepSz<T> src, PtrStepb dst)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
        const int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

        if (x + 1 >= src.cols || y + 1 >= src.rows)
            return;

        // get pointers to the data
        const size_t planeSize = src.rows * dst.step;
        PtrStepb y_plane(dst.data, dst.step);
        PtrStepb u_plane(y_plane.data + planeSize, dst.step / 2);
        PtrStepb v_plane(u_plane.data + (planeSize / 4), dst.step / 2);

        T pix;
        uchar y_val, u_val, v_val;

        pix = src(y, x);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y, x) = y_val;

        pix = src(y, x + 1);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y, x + 1) = y_val;

        pix = src(y + 1, x);
        rgb_to_y(pix.z, pix.y, pix.x, y_val);
        y_plane(y + 1, x) = y_val;

        pix = src(y + 1, x + 1);
        rgb_to_yuv(pix.z, pix.y, pix.x, y_val, u_val, v_val);
        y_plane(y + 1, x + 1) = y_val;
        u_plane(y / 2, x / 2) = u_val;
        v_plane(y / 2, x / 2) = v_val;
    }

    void Gray_to_YV12_caller(const PtrStepSzb src, PtrStepb dst, hipStream_t stream)
    {
        dim3 block(32, 8);
        dim3 grid(divUp(src.cols, block.x * 2), divUp(src.rows, block.y * 2));

        Gray_to_YV12<<<grid, block, 0, stream>>>(src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
    template <int cn>
    void RGB_to_YV12_caller(const PtrStepSzb src, PtrStepb dst, hipStream_t stream)
    {
        typedef typename TypeVec<uchar, cn>::vec_type src_t;

        dim3 block(32, 8);
        dim3 grid(divUp(src.cols, block.x * 2), divUp(src.rows, block.y * 2));

        RGB_to_YV12<<<grid, block, 0, stream>>>(static_cast< PtrStepSz<src_t> >(src), dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void RGB_to_YV12(const PtrStepSzb src, int cn, PtrStepSzb dst, hipStream_t stream)
    {
        typedef void (*func_t)(const PtrStepSzb src, PtrStepb dst, hipStream_t stream);

        static const func_t funcs[] =
        {
            0, Gray_to_YV12_caller, 0, RGB_to_YV12_caller<3>, RGB_to_YV12_caller<4>
        };

        funcs[cn](src, dst, stream);
    }
}}}
